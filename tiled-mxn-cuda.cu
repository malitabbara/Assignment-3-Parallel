// Tiled matrix multiplication
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 16

// CUDA kernel function
__global__ void matrix_multiplication_tiled(float *A, float *B, float *C, int M, int N) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < M && j < N) {
    float sum = 0.0f;
    for (int k = 0; k < N; k += TILE_SIZE) {
      sum += A[i * N + k] * B[k * N + j];
    }
    C[i * N + j] = sum;
  }
}

// Main function
int main() {
  // Declare variables
  int M = 1024;
  int N = 1024;
  int rep = 0;
  float *A, *B, *C;
while(rep<100){
  // Allocate memory on the host
  A = (float *)malloc(sizeof(float) * M * N);
  B = (float *)malloc(sizeof(float) * N * N);
  C = (float *)malloc(sizeof(float) * M * N);

  // Fill the matrices with random numbers
  for (int i = 0; i < M * N; i++) {
    A[i] = rand() / (float)RAND_MAX;
  }
  for (int i = 0; i < N * N; i++) {
    B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate memory on the device
  float *d_A, *d_B, *d_C;
  hipMalloc((void **)&d_A, sizeof(float) * M * N);
  hipMalloc((void **)&d_B, sizeof(float) * N * N);
  hipMalloc((void **)&d_C, sizeof(float) * M * N);

  // Copy the matrices to the device
  hipMemcpy(d_A, A, sizeof(float) * M * N, hipMemcpyHostToDevice);
  hipMemcpy(d_B, B, sizeof(float) * N * N, hipMemcpyHostToDevice);

  // Launch the kernel
  dim3 dimBlock(16, 16);
  dim3 dimGrid(M / dimBlock.x, N / dimBlock.y);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  matrix_multiplication_tiled<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, M, N);
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, start, stop);

  // Copy the result back to the host
  hipMemcpy(C, d_C, sizeof(float) * M * N, hipMemcpyDeviceToHost);

  // Print the time taken for execution
  printf("%f ,\n", elapsedTime);

  // Free memory
  free(A);
  free(B);
  free(C);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
  rep++;
}
  return 0;
}

